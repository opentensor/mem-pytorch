#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <c10/cuda/CUDAGuard.h>

#include <torch/extension.h>

// error handler
// from https://leimao.github.io/blog/Proper-CUDA-Error-Checking

#define CHECK_LAST_CUDA_ERROR() check(__FILE__, __LINE__)
void check(const char* file, const int line)
{
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        std::cerr << "CUDA Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}

#define ACCESSOR(x, n, type) x.packed_accessor32<type, n, torch::RestrictPtrTraits>()

// type alias

template <typename scalar_t, int dims>
using PackedAccessor = torch::PackedTensorAccessor32<scalar_t, dims, torch::RestrictPtrTraits>;

// helper functions

__host__ __device__ int cdiv(int numer, int denom) {
    return (numer + denom - 1) / denom;
}

__host__ __device__ int next_multiple_of(int num, int multiple_of) {
    return cdiv(num, multiple_of) * multiple_of;
}

__host__ __device__ int next_pow_2(int n) {
    int i = 1;
    while(i < n)
        i *= 2;
    return i;
}

__device__ void warp_reduce(volatile float* sm, int tid, int max) {
    for (int s = 32; s > 0; s>>=1) {
        if ((tid + s) >= max)
            continue;

        sm[tid] += sm[tid + s];
    }
}

bool divisible_by(int num, int denom) {
    return (num % denom) == 0;
}

// forward kernel

template <typename scalar_t>
__global__ void forward_kernel(
    const PackedAccessor<scalar_t, 4> q,
    const PackedAccessor<scalar_t, 4> k,
    const PackedAccessor<scalar_t, 4> v,
    const PackedAccessor<bool, 2> mask,
    const PackedAccessor<scalar_t, 3> attn_bias,
          PackedAccessor<scalar_t, 4> o,
          PackedAccessor<scalar_t, 3> l,
    const float scale,
    const bool causal,
    const bool has_mask,
    const bool has_attn_bias,
    const int row_tile_size,
    const int col_tile_size,
    const int col_tiles
) {
    const int batch = q.size(0);
    const int head = q.size(1);

    const int batch_idx = blockIdx.x / head;
    const int head_idx = blockIdx.x % head;

    const int q_seq_len = q.size(2);
    const int k_seq_len = k.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    const int num_col_tiles = cdiv(k_seq_len, col_tile_size);

    const int row_tiles_idx = blockIdx.y / col_tiles;
    const int col_tiles_idx = blockIdx.y % col_tiles;

    const int col_tile_idx = threadIdx.x;
    const int row_tile_idx = threadIdx.y;

    // for coalesced reads

    const int thread_idx = col_tile_size * row_tile_idx + col_tile_idx;
    const int tpb = blockDim.x * blockDim.y;

    // for warp reducing

    const int lane_id = threadIdx.x & 31;
    const int col_num_warps = col_tile_size / 32;
    const int warp_id_per_row = col_tile_idx / 32;

    const int sm_q_offset = row_tile_idx * k_dim;
    const int sm_k_offset = col_tile_idx * k_dim;
    const int sm_v_offset = col_tile_idx * v_dim;
    const int sm_o_offset = row_tile_idx * v_dim;

    const int k_total_el = col_tile_size * k_dim;
    const int v_total_el = col_tile_size * v_dim;

    auto q_ = q[batch_idx][head_idx];
    auto k_ = k[batch_idx][head_idx];
    auto v_ = v[batch_idx][head_idx];
    auto o_ = o[batch_idx][head_idx];
    auto l_ = l[batch_idx][head_idx];
    auto mask_ = mask[batch_idx];

    // handle attention bias

    auto attn_bias_ = has_attn_bias ? attn_bias[head_idx] : attn_bias[0];

    // shared memory

    extern __shared__ float _shared_mem[];

    float* sm_q = (float*) &_shared_mem;
    float* sm_k = (float*) &sm_q[row_tile_size * k_dim];
    float* sm_v = (float*) &sm_k[col_tile_size * k_dim];
    float* sm_l = (float*) &sm_v[col_tile_size * v_dim];

    // some variable

    int col_tiles_offset;
    int global_col;
    bool should_calculate_attn, should_calculate_col;

    const int row_tiles_offset = row_tiles_idx * row_tile_size;
    const int global_row = row_tiles_offset + row_tile_idx;
    const bool should_calculate_row = global_row < q_seq_len;

    // loop

    for (
        int d = row_tile_idx;
        d < k_dim;
        d += row_tile_size
    ) {
        sm_q[col_tile_idx * k_dim + d] = q_[row_tiles_offset + col_tile_idx][d];
    }

    float acc_attn = 0;

    for (int i = 0; i < num_col_tiles; i++) {
        col_tiles_offset = i * col_tile_size;
        global_col = col_tiles_offset + col_tiles_idx * col_tile_size + col_tile_idx;
        should_calculate_col = global_col < k_seq_len && (!has_mask || mask_[global_col]);

        // coalesced reads from hbm
        // cleanup later, make it work first

        for (
            int offset = 0;
            offset < k_total_el;
            offset += tpb
        ) {
            int smem_idx = offset + thread_idx;
            int gmem_seq_idx = smem_idx / k_dim;
            int gmem_dim_idx = smem_idx % k_dim;

            if (smem_idx < k_total_el)
                sm_k[smem_idx] = k_[col_tiles_offset + col_tiles_idx * col_tile_size + gmem_seq_idx][gmem_dim_idx];
        }

        for (
            int offset = 0;
            offset < v_total_el;
            offset += tpb
        ) {
            int smem_idx = offset + thread_idx;
            int gmem_seq_idx = smem_idx / v_dim;
            int gmem_dim_idx = smem_idx % v_dim;

            if (smem_idx < v_total_el)
                sm_v[smem_idx] = v_[col_tiles_offset + col_tiles_idx * col_tile_size + gmem_seq_idx][gmem_dim_idx];
        }

        __syncthreads();

        should_calculate_attn = should_calculate_row &&
                                should_calculate_col &&
                                ( !causal ||
                                  (causal && (global_row >= (global_col - k_seq_len + q_seq_len))));

        float attn = 0;

        if (should_calculate_attn) {
            for (int d = 0; d < k_dim; d++) {
                // dmod is a "hacky" way to avoid bank register conflicts from @ahennequ
                int dmod = (d + lane_id) % k_dim;
                attn += sm_q[sm_q_offset + dmod] * sm_k[sm_k_offset + dmod];
            }

            attn *= scale;

            if (has_attn_bias) {
                attn += attn_bias_[global_row][global_col];
            }

            attn -= scale;
            attn = __expf(attn);

            float exp_weighted_value;

            for (int d = 0; d < v_dim; d++) {
                exp_weighted_value = attn * sm_v[sm_v_offset + d];
                atomicAdd((float*) &o_[global_row][d], exp_weighted_value);
            }
        }

        __syncthreads();

        acc_attn += attn;

    }

    // reduce accumulated attention from inner loop

    const unsigned shfl_mask = __ballot_sync(0xFFFFFFFFU, should_calculate_row);

    for (int offset = 16; offset > 0; offset >>= 1) {
        acc_attn += __shfl_down_sync(shfl_mask, acc_attn, offset);
    }

    if (lane_id == 0 && warp_id_per_row > 0)
        sm_l[row_tile_idx * (col_num_warps - 1)  + (warp_id_per_row - 1)] = acc_attn;

    __syncthreads();

    if (warp_id_per_row == 0) {
        // if not the first column, and column is less than the number of warps per row
        // then get from shared memory, set everything else but the first column to 0.

        if (col_tile_idx > 0 && col_tile_idx < col_num_warps) {
            acc_attn = sm_l[row_tile_idx * col_num_warps + (col_tile_idx - 1)];
        } else if (col_tile_idx != 0) {
            acc_attn = 0;
        }

        for (int offset = 16; offset > 0; offset >>= 1) {
            acc_attn += __shfl_down_sync(shfl_mask, acc_attn, offset);
        }
    }

    // write row sum (accumulated by thread of first column per row) to global memory

    if (should_calculate_row && col_tile_idx == 0)
        l_[global_row] = acc_attn;
}

// forwards c++ function

std::vector<torch::Tensor> flash_cosine_sim_attention_forward(
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    torch::Tensor mask,
    torch::Tensor attn_bias,
    float scale,
    bool causal,
    int row_tile_size,
    int col_tile_size,
    int col_tiles
) {

    auto query_device = device_of(q);
    const at::cuda::OptionalCUDAGuard device_guard(query_device);

    const int batch = q.size(0);
    const int heads = q.size(1);
    const int seq   = q.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    auto options = torch::TensorOptions().device(query_device).dtype(torch::kFloat);

    auto o = at::zeros({batch, heads, seq, v_dim}, options);
    auto l = at::zeros({batch, heads, seq}, options);

    const bool has_attn_bias = !!attn_bias.numel();
    const bool has_mask = !!mask.numel();

    const int row_tiles = cdiv(seq, row_tile_size);

    const dim3 blocks(batch * heads, row_tiles * col_tiles);
    const dim3 threads_per_block(col_tile_size, row_tile_size);

    const unsigned shared_mem_size = (
                                        row_tile_size * k_dim +                       // q
                                        col_tile_size * k_dim +                       // k
                                        col_tile_size * v_dim +                       // v
                                        ((col_tile_size / 32) - 1) * row_tile_size    // l per (warps - 1)
                                     ) * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(q.scalar_type(), "forward_cosine_sim_attention_forward", ([&] {
        forward_kernel<scalar_t><<<blocks, threads_per_block, shared_mem_size>>>(
            ACCESSOR(q, 4, scalar_t),
            ACCESSOR(k, 4, scalar_t),
            ACCESSOR(v, 4, scalar_t),
            ACCESSOR(mask, 2, bool),
            ACCESSOR(attn_bias, 3, scalar_t),
            ACCESSOR(o, 4, scalar_t),
            ACCESSOR(l, 3, scalar_t),
            scale,
            causal,
            has_mask,
            has_attn_bias,
            row_tile_size,
            col_tile_size,
            col_tiles
        );
    }));

    hipDeviceSynchronize();

    // handle error

    CHECK_LAST_CUDA_ERROR();

    return {o, l};
}

// backward kernel

// backwards preprocess

// calculate do_scaled = rowsum(do * o)
// done by @ptillet at https://github.com/openai/triton/blob/master/python/tutorials/06-fused-attention.py

template <typename scalar_t>
__global__ void backward_calculate_do_scaled(
    const PackedAccessor<scalar_t, 4> d_out,
    const PackedAccessor<scalar_t, 4> o,
          PackedAccessor<scalar_t, 3> do_scaled
) {
    const int heads = o.size(1);
    const int v_dim = o.size(3);

    const int batch_idx = blockIdx.x / heads;
    const int head_idx = blockIdx.x % heads;
    const int seq_idx = blockIdx.y;
    const int dim_idx = threadIdx.x;

    const int warp_id = threadIdx.x / 32;
    const int lane_id = threadIdx.x & 31;

    const unsigned mask = __ballot_sync(0xFFFFFFFFU, dim_idx < v_dim);

    float val = 0.0f;

    extern __shared__ float _shared_mem_preprocess[];

    float* sm_do_scaled = (float*) &_shared_mem_preprocess;

    auto do_ = d_out[batch_idx][head_idx][seq_idx];
    auto o_ = o[batch_idx][head_idx][seq_idx];
    auto do_scaled_ = do_scaled[batch_idx][head_idx];

    // load into shared memory

    if (dim_idx < v_dim)
        val = do_[dim_idx] * o_[dim_idx];

    // warp shuffle reduce

    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(mask, val, offset);
    }

    if (lane_id == 0)
        sm_do_scaled[warp_id] = val;

    __syncthreads();

    if (warp_id == 0) {
        if (dim_idx < (blockDim.x / 32)) {
            val = sm_do_scaled[lane_id];
        } else{
            val = 0;
        }

        for (int offset = 16; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(mask, val, offset);
        }

        if (dim_idx == 0) {
            do_scaled_[seq_idx] = val;
        }
    }
}

// main backward kernel

template <typename scalar_t>
__global__ void backward_kernel(
    const PackedAccessor<scalar_t, 4> q,
    const PackedAccessor<scalar_t, 4> k,
    const PackedAccessor<scalar_t, 4> v,
    const PackedAccessor<bool, 2> mask,
    const PackedAccessor<scalar_t, 3> attn_bias,
          PackedAccessor<scalar_t, 4> dq,
          PackedAccessor<scalar_t, 4> dk,
          PackedAccessor<scalar_t, 4> dv,
          PackedAccessor<scalar_t, 3> d_attn_bias,
    const PackedAccessor<scalar_t, 4> d_out,
    const PackedAccessor<scalar_t, 3> do_scaled,
    const PackedAccessor<scalar_t, 3> l,
    const float scale,
    const bool causal,
    const bool has_mask,
    const bool has_attn_bias,
    const int row_tile_size,
    const int col_tile_size,
    const int row_tiles,
    const int col_tiles
) {

    const int batch = q.size(0);
    const int head = q.size(1);

    const int batch_idx = blockIdx.x / head;
    const int head_idx = blockIdx.x % head;

    const int q_seq_len = q.size(2);
    const int k_seq_len = k.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    const int num_col_tiles = cdiv(k_seq_len, col_tile_size);
    const int num_row_tiles = cdiv(q_seq_len, row_tile_size);

    const int row_tiles_idx = blockIdx.y / col_tiles;
    const int col_tiles_idx = blockIdx.y % col_tiles;

    const int col_tile_idx = threadIdx.x;
    const int row_tile_idx = threadIdx.y;

    const int lane_id = threadIdx.x & 31;

    const int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
    const int tpb = blockDim.x * blockDim.y;

    const int k_total_el = k_dim * col_tile_size;
    const int v_total_el = v_dim * col_tile_size;

    const int sm_q_offset = row_tile_idx * k_dim;
    const int sm_k_offset = col_tile_idx * k_dim;
    const int sm_v_offset = col_tile_idx * v_dim;
    const int sm_o_offset = row_tile_idx * v_dim;

    auto q_ = q[batch_idx][head_idx];
    auto k_ = k[batch_idx][head_idx];
    auto v_ = v[batch_idx][head_idx];
    auto dq_ = dq[batch_idx][head_idx];
    auto dk_ = dk[batch_idx][head_idx];
    auto dv_ = dv[batch_idx][head_idx];
    auto ds_ = d_attn_bias[head_idx];
    auto do_scaled_ = do_scaled[batch_idx][head_idx];
    auto l_ = l[batch_idx][head_idx];
    auto do_ = d_out[batch_idx][head_idx];
    auto mask_ = mask[batch_idx];

    // handle attention bias

    auto attn_bias_ = has_attn_bias ? attn_bias[head_idx] : attn_bias[0];

    // some variables

    int col_tiles_offset, row_tiles_offset;
    int global_col, global_row;
    bool should_calculate_attn, should_calculate_row, should_calculate_col;

    // shared memory

    extern __shared__ float _shared_mem[];

    float* sm_q = (float*) &_shared_mem;
    float* sm_k = (float*) &sm_q[row_tile_size * k_dim];
    float* sm_v = (float*) &sm_k[col_tile_size * k_dim];
    float* sm_l = (float*) &sm_v[col_tile_size * v_dim];
    float* sm_do_scaled = (float*) &sm_l[row_tile_size];
    float* sm_do = (float*) &sm_do_scaled[row_tile_size];

    // loop

    for (int i = 0; i < num_col_tiles; i++) {
        col_tiles_offset = i * col_tile_size;
        global_col = col_tiles_offset + col_tiles_idx * col_tile_size + col_tile_idx;
        should_calculate_col = global_col < k_seq_len && (!has_mask || mask_[global_col]);

        // coalesced reads
        // cleanup later

        for (
            int offset = 0;
            offset < k_total_el;
            offset += tpb
        ) {
            int sm_idx = offset + thread_idx;
            int gmem_seq_idx = sm_idx / k_dim;
            int gmem_dim_idx = sm_idx % k_dim;

            if (offset < k_total_el)
                sm_k[sm_idx] = k_[col_tiles_offset + col_tiles_idx * col_tile_size + gmem_seq_idx][gmem_dim_idx];
        }

        for (
            int offset = 0;
            offset < v_total_el;
            offset += tpb
        ) {
            int sm_idx = offset + thread_idx;
            int gmem_seq_idx = sm_idx / v_dim;
            int gmem_dim_idx = sm_idx % v_dim;

            if (offset < v_total_el)
                sm_v[sm_idx] = v_[col_tiles_offset + col_tiles_idx * col_tile_size + gmem_seq_idx][gmem_dim_idx];
        }

        for (int j = 0; j < num_row_tiles; j++) {
            row_tiles_offset = j * row_tile_size;
            global_row = row_tiles_offset + row_tiles_idx * row_tile_size + row_tile_idx;
            should_calculate_row = global_row < q_seq_len;

            should_calculate_attn = should_calculate_row &&
                                    should_calculate_col &&
                                    ( !causal ||
                                      (causal && (global_row >= (global_col - k_seq_len + q_seq_len))));

            for (
                int d = col_tile_idx;
                d < k_dim;
                d += col_tile_size
            ) {
                sm_q[row_tile_idx * k_dim + d] = q_[row_tiles_offset + row_tiles_idx * row_tile_size + row_tile_idx][d];
            }

            for (
                int d = col_tile_idx;
                d < v_dim;
                d += col_tile_size
            ) {
                sm_do[row_tile_idx * v_dim + d] = do_[row_tiles_offset + row_tiles_idx * row_tile_size + row_tile_idx][d];
            }

            if (col_tile_idx == 0) {
                sm_do_scaled[row_tile_idx] = do_scaled_[global_row];
                sm_l[row_tile_idx] = l_[global_row];
            }

            __syncthreads();

            float attn = 0;
            float row_sum = 0;
            float dp = 0;

            if (should_calculate_attn) {
                for (int d = 0; d < k_dim; d++) {
                    // dmod is a "hacky" way to avoid bank register conflicts from @ahennequ
                    int dmod = (d + lane_id) % k_dim;
                    attn += sm_q[sm_q_offset + dmod] * sm_k[sm_k_offset + dmod];
                }

                attn *= scale;

                if (has_attn_bias) {
                    attn += attn_bias_[global_row][global_col];
                }

                attn -= scale;
                attn = __expf(attn);

                row_sum = sm_l[row_tile_idx];

                if (row_sum > 1e-8)
                    attn /= row_sum;

                for (int d = 0; d < v_dim; d++) {
                    // accumulate dv to global mem

                    atomicAdd((float*) &dv_[global_col][d], sm_do[sm_o_offset + d] * attn);

                    // calculate dp

                    dp += sm_do[sm_o_offset + d] * sm_v[sm_v_offset + d];
                }
            }

            // calculate dS

            float dS = 0;

            if (should_calculate_attn) {
                float D = sm_do_scaled[row_tile_idx];

                dS = attn * (dp - D);

                if (has_attn_bias) {
                    atomicAdd((float*) &ds_[global_row][global_col], dS);
                }
            }

            __syncthreads();

            // accumulate dq and dk to global mem

            if (should_calculate_attn) {
                dS *= scale;

                for (int d = 0; d < k_dim; d++) {
                    atomicAdd((float*) &dq_[global_row][d], dS * sm_k[sm_k_offset + d]);

                    atomicAdd((float*) &dk_[global_col][d], dS * sm_q[sm_q_offset + d]);
                }
            }

            __syncthreads();
        }
    }
}

// backwards c++ function

std::vector<torch::Tensor> flash_cosine_sim_attention_backward(
    torch::Tensor d_out,
    torch::Tensor o,
    torch::Tensor l,
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    torch::Tensor d_attn_bias,
    torch::Tensor mask,
    torch::Tensor attn_bias,
    float scale,
    bool causal,
    int row_tile_size,
    int col_tile_size,
    int row_tiles,
    int col_tiles
) {
    auto query_device = device_of(q);

    const at::cuda::OptionalCUDAGuard device_guard(query_device);

    const int batch = q.size(0);
    const int heads = q.size(1);
    const int seq   = q.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    const bool has_attn_bias = !!d_attn_bias.numel();
    const bool has_mask = !!mask.numel();

    auto options = torch::TensorOptions().device(query_device).dtype(torch::kFloat);

    // setup dq, dk, dv

    auto do_scaled = at::empty_like(l, options);

    auto dq = at::zeros_like(q, options);
    auto dk = at::zeros_like(k, options);
    auto dv = at::zeros_like(v, options);

    // setup backwards preprocess call

    const dim3 backwards_preprocess_threads_per_block(next_multiple_of(v_dim, 32));

    const dim3 backwards_preprocess_blocks(batch * heads, seq);

    const unsigned backwards_preprocess_shared_mem_size = cdiv(v_dim, 32) * sizeof(float);

    // setup backwards call

    const dim3 backwards_threads_per_block(col_tile_size, row_tile_size);
    const dim3 backwards_blocks(batch * heads, row_tiles * col_tiles);

    const unsigned backwards_shared_mem_size = (  (row_tile_size + col_tile_size) * k_dim +      // q, k
                                                  (row_tile_size + col_tile_size) * v_dim +      // v, do
                                                  (row_tile_size + col_tile_size)                // l, do_scaled
                                                ) * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(q.scalar_type(), "forward_cosine_sim_attention_backward", ([&] {
        backward_calculate_do_scaled<scalar_t><<<backwards_preprocess_blocks, backwards_preprocess_threads_per_block, backwards_preprocess_shared_mem_size>>>(
            ACCESSOR(d_out, 4, scalar_t),
            ACCESSOR(o, 4, scalar_t),
            ACCESSOR(do_scaled, 3, scalar_t)
        );

        backward_kernel<scalar_t><<<backwards_blocks, backwards_threads_per_block, backwards_shared_mem_size>>>(
            ACCESSOR(q, 4, scalar_t),
            ACCESSOR(k, 4, scalar_t),
            ACCESSOR(v, 4, scalar_t),
            ACCESSOR(mask, 2, bool),
            ACCESSOR(attn_bias, 3, scalar_t),
            ACCESSOR(dq, 4, scalar_t),
            ACCESSOR(dk, 4, scalar_t),
            ACCESSOR(dv, 4, scalar_t),
            ACCESSOR(d_attn_bias, 3, scalar_t),
            ACCESSOR(d_out, 4, scalar_t),
            ACCESSOR(do_scaled, 3, scalar_t),
            ACCESSOR(l, 3, scalar_t),
            scale,
            causal,
            has_mask,
            has_attn_bias,
            row_tile_size,
            col_tile_size,
            row_tiles,
            col_tiles
        );
    }));

    hipDeviceSynchronize();

    // handle error

    CHECK_LAST_CUDA_ERROR();

    return {dq, dk, dv};
}

// bind

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &flash_cosine_sim_attention_forward, "Flash Cosine-Sim Attention Forward");
    m.def("backward", &flash_cosine_sim_attention_backward, "Flash Cosine-Sim Attention Backward");
}